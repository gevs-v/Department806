#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

__global__ void vectorSqrt(const float* A, float* C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = sqrtf(A[i]);
    }
}

int
main(void)
{
    // ���� �� ����������� ������ ��������
    int pow = 6;
    int numElements = 100;
    while (numElements <= numElements * pow)
    {
        // �������� ������� ��� �������� ����� ���������� �������
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // ��������� ������ ��� ������� �������� �����
        size_t size = numElements * sizeof(float);

        float* h_A = (float*)malloc(size);
        float* h_C = (float*)malloc(size);

        if (h_A == NULL || h_C == NULL)
        {
            fprintf(stderr, "Failed to allocate host vectors!\n");
            exit(EXIT_FAILURE);
        }

        // ������������� ���������� ���������� ������������ �������
        for (int i = 0; i < numElements; ++i)
        {
            h_A[i] = rand() / (float)RAND_MAX;
        }

        // ����� ������� �� ����������� ������ �� ����������, ���������� ������� � ����������� ���������� �������
        hipEventRecord(start);
        
        // ��������� �������� �������� ����� �� ����������
        float* d_A = NULL;
        hipMalloc((void**)&d_A, size);
        float* d_C = NULL;
        hipMalloc((void**)&d_C, size);

        // ����������� ������� � ����� �� ����������
        hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

        // ��������� ������ � ������
        int threadsPerBlock = 256;
        int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

        // ������ ���������� �������
        vectorSqrt <<<blocksPerGrid, threadsPerBlock >>> (d_A, d_C, numElements);

        // ����������� ����������� ������� �� ����
        hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
        hipEventRecord(stop);

        // ��������� �������
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        printf("For % d elements time elapsed in ms : % f\n", numElements, milliseconds);

        // ������������ ������ �� ����� � �� ����������
        hipFree(d_A);
        hipFree(d_C);

        free(h_A);
        free(h_C);

        numElements *= 10;
    }
    
    return 0;
}

